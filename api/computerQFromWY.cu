
#include "computerQFromWY.h"
#include "kernelOther.h"

// Q = I - WY'
// W和Y的维度MxN
// Q的维度是MxM
void computerQFromWY(hipblasHandle_t cublas_handle, long M, long N, double *dQ, long ldQ,
                     double *dW, long ldW, double *dY, long ldY)
{
    double done = 1.0;
    double dzero = 0.0;

    // 计算WY'
    hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, M, M, N,
                &done, dW, ldW, dY, ldY, &dzero, dQ, ldQ);

    // 计算I-WY'
    dim3 gridDim((M + 31) / 32, (M + 31) / 32);
    dim3 blockDim(32, 32);
    launchKernel_IminusQ(gridDim, blockDim, M, M, dQ, ldQ);
}