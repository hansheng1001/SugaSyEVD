
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>

#include <hiprand/hiprand.h>

// 定义函数模版
template <typename T>
void generateUniformMatrix(T *dA, long int m, long int n);

// 特例化模版函数
template <>
void generateUniformMatrix(double *dA, long int m, long int n)
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  int seed = 3000;
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  hiprandGenerateUniformDouble(gen, dA, long(m * n));
}

template <>
void generateUniformMatrix(float *dA, long int m, long int n)
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  int seed = 3000;
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniform(gen, dA, long(m * n));
}

__global__ static void
matrixCpyF2H(long int m, long int n, float *a, long int lda, half *b, long int ldb)
{
  long int i = threadIdx.x + blockDim.x * blockIdx.x;
  long int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i < m && j < n)
  {
    b[i + j * ldb] = __half2float(a[i + j * lda]);
  }
}

template <>
void generateUniformMatrix(half *dA, long int m, long int n)
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  int seed = 3000;
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  float *_dA;
  hipMalloc((void **)&_dA, sizeof(float) * m * n);

  hiprandGenerateUniform(gen, _dA, long(m * n));

  dim3 blockDim(32, 32);
  dim3 gridDim((m + 32 - 1) / 32, (n + 32 - 1) / 32);
  matrixCpyF2H<<<gridDim, blockDim>>>(m, n, _dA, m, dA, m);

  hipFree(_dA);
}

template <typename T>
void generateNormalMatrix(T *dA, long int m, long int n, T mean, T stddev);

template <>
void generateNormalMatrix(double *dA, long int m, long int n, double mean, double stddev)
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  int seed = 3000;
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  hiprandGenerateNormalDouble(gen, dA, long(m * n), mean, stddev);
}

template <>
void generateNormalMatrix(float *dA, long int m, long int n, float mean, float stddev)
{
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  int seed = 3000;
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  hiprandGenerateNormal(gen, dA, long(m * n), mean, stddev);
}

template <typename T>
void printDeviceMatrixV2(T *dA, long ldA, long rows, long cols)
{
  T matrix;

  for (long i = 0; i < rows; i++)
  {
    for (long j = 0; j < cols; j++)
    {
      hipMemcpy(&matrix, dA + i + j * ldA, sizeof(T), hipMemcpyDeviceToHost);
      // printf("%f ", matrix[i * cols + j]);//按行存储优先
      // printf("%10.4f", matrix); // 按列存储优先
      // printf("%12.6f", matrix); // 按列存储优先
      // printf("%.20f ", matrix); // 按列存储优先
      printf("%.14f ", matrix); // 按列存储优先
    }
    printf("\n");
  }
}

template void printDeviceMatrixV2(double *dA, long ldA, long rows, long cols);
template void printDeviceMatrixV2(float *dA, long ldA, long rows, long cols);

template <>
void printDeviceMatrixV2(half *dA, long ldA, long rows, long cols)
{
  half matrix;

  for (long i = 0; i < rows; i++)
  {
    for (long j = 0; j < cols; j++)
    {
      hipMemcpy(&matrix, dA + i + j * ldA, sizeof(half), hipMemcpyDeviceToHost);
      float f = __half2float(matrix);
      // printf("%f ", matrix[i * cols + j]);//按行存储优先
      // printf("%10.4f", matrix); // 按列存储优先
      // printf("%12.6f", matrix); // 按列存储优先
      // printf("%.20f ", matrix); // 按列存储优先
      printf("%.14f ", f); // 按列存储优先
    }
    printf("\n");
  }
}

std::vector<std::vector<double>> readMatrixFromFile(const std::string &fileName)
{
  std::vector<std::vector<double>> matrix;

  std::ifstream file(fileName);

  if (file.is_open())
  {
    std::string line;
    while (getline(file, line))
    {
      std::vector<double> row;
      std::stringstream ss(line);
      std::string cell;

      while (getline(ss, cell, ','))
      {
        row.push_back(std::stod(cell));
      }

      matrix.push_back(row);
    }

    file.close();
    std::cout << "Matrix read from " << fileName << std::endl;
  }
  else
  {
    std::cout << "Failed to open file: " << fileName << std::endl;
  }

  return matrix;
}

void fillMatrix(double *matrix, std::vector<std::vector<double>> &data)
{
  long rows = data.size();
  long cols = data[0].size();

  // 这是行优先(row-major order, RMO)的存储方式
  // for (long i = 0; i < rows; i++) {
  //   for (long j = 0; j < cols; j++) {
  //     matrix[i * cols + j] = data[i][j];
  //   }
  // }

  for (long i = 0; i < cols; i++)
  {
    for (long j = 0; j < rows; j++)
    {
      matrix[i * rows + j] = data[j][i];
    }
  }
}

void printMatrix(double *matrix, long ldA, long rows, long cols)
{
  for (long i = 0; i < rows; i++)
  {
    for (long j = 0; j < cols; j++)
    {
      // printf("%f ", matrix[i * cols + j]);//按行存储优先
      printf("%0.14f ", matrix[j * ldA + i]); // 按列存储优先
    }
    printf("\n");
  }
}

void printDeviceMatrixV2Int(int *dA, long ldA, long rows, long cols)
{
  int matrix;

  for (long i = 0; i < rows; i++)
  {
    for (long j = 0; j < cols; j++)
    {
      hipMemcpy(&matrix, dA + i + j * ldA, sizeof(int), hipMemcpyDeviceToHost);
      // printf("%f ", matrix[i * cols + j]);//按行存储优先
      // printf("%10.4f", matrix); // 按列存储优先
      // printf("%12.6f", matrix); // 按列存储优先
      printf("%d ", matrix); // 按列存储优先
    }
    printf("\n");
  }
}

// template<typename T>
// void printDeviceMatrix(T *dA, long rows, long cols) {

//   T *matrix;
//   matrix = (T *)malloc(sizeof(T) * m * n);

//   cudaMemcpy(matrix, dA, sizeof(T) * m * n, cudaMemcpyDeviceToHost);

//   for (long i = 0; i < rows; i++) {
//     for (long j = 0; j < cols; j++) {
//       // printf("%f ", matrix[i * cols + j]);//按行存储优先
//       printf("%10.4f", matrix[j * rows + i]);  // 按列存储优先
//     }
//     printf("\n");
//   }

//   free(matrix);
// }